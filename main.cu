#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

//лучший результат при соотношении 1 к 25
#define N 2000
#define div 80

//алгоритм Флойда-Уоршелла
__global__ void floyd(int* b, int i) {
	int k = blockIdx.x*(N / div) + threadIdx.x;
	int j = blockIdx.y*(N / div) + threadIdx.y;
	if (b[j * N + k] > b[j * N + i] + b[i * N + k]) {
		b[j * N + k] = b[j * N + i] + b[i * N + k];
	}
}

int main()
{
	//заполнение матрицы смежности
	int* G;
	G = new int[N * N];
	for (int i = 0; i < N; ++i) {
		for (int j = i; j < N; ++j) {
			if (i == j) {
				G[i * N + j] = 0;
			}
			else {
				G[i * N + j] = G[j * N + i] = rand() % 10;
			}
		}
	}

	printf("N = %d \n", N);
	
	/*printf("\n INPUT: \n");

	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", G[i * N + j]);
		}
		printf("\n");
	}*/

	int * dev;
	hipMalloc((void**)&dev, N * N * sizeof(int));

	hipError_t error;
	error = hipMemcpy(dev, G, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("%s\n", hipGetErrorString(error));
	}

	dim3 grid(div, div);
	dim3 blocks(N/div, N/div);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	//запускаем алгоритм
	for (int i = 0; i < N; ++i) {
		floyd << <grid, blocks >> > (dev, i);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	//записываем время работы
	float time = 0;
	hipEventElapsedTime(&time, start, stop);

	error = hipGetLastError();
	if (error != hipSuccess){
		printf("%s\n", hipGetErrorString(error));
	}

	hipDeviceSynchronize();

	error = hipMemcpy(G, dev, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("%s\n", hipGetErrorString(error));
	}

	/*printf("\n RESULT: \n");

	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%d ", G[i * N + j]);
		}
		printf("\n");
	}*/

	printf("\nTIME: \n");
	printf("%f ms\n", time);

	delete G;
	hipFree(dev);
	return 0;
}